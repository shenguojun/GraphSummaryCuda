#include "init.h"
#include "graph.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void cudaMemoryHostToDeviceMalloc(Graph* pDevicGraph, Graph* pHostGraph){
	int nVerNum = pHostGraph->m_nVerNum;
	int nEdgeNum = pHostGraph->m_nEdgeNum;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	size_t unVerSize = nVerNum * sizeof(int);
	size_t unVerSizePlus = (nVerNum + 1) * sizeof(int);
	size_t unEdgeSize = nEdgeNum * sizeof(int);
	// Allocate GPU buffers for graph vectors
	hipMalloc((void**)&pDevicGraph->m_anOwnerId, unVerSize);
	hipMalloc((void**)&pDevicGraph->m_anSelfConn, unVerSize);
	hipMalloc((void**)&pDevicGraph->m_anVrtexSize, unVerSize);
	hipMalloc((void**)&pDevicGraph->m_anRowOffset, unVerSizePlus);
	hipMalloc((void**)&pDevicGraph->m_anNeighborId, unEdgeSize);
	hipMalloc((void**)&pDevicGraph->m_anBetweenConn, unEdgeSize);


	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(pDevicGraph->m_anOwnerId, pHostGraph->m_anOwnerId, nVerNum * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pDevicGraph->m_anSelfConn, pHostGraph->m_anSelfConn, nVerNum * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pDevicGraph->m_anVrtexSize, pHostGraph->m_anVrtexSize, nVerNum * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pDevicGraph->m_anRowOffset, pHostGraph->m_anRowOffset, (nVerNum + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pDevicGraph->m_anNeighborId, pHostGraph->m_anNeighborId, nEdgeNum * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pDevicGraph->m_anBetweenConn, pHostGraph->m_anBetweenConn, nEdgeNum * sizeof(int), hipMemcpyHostToDevice);

}