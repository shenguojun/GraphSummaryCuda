#include "init.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void cudaMemoryHostToDeviceMalloc(Graph* pDevicGraph, Graph* pHostGraph){
	int nVerNum = pHostGraph->m_nVerNum;
	int nEdgeNum = pHostGraph->m_nEdgeNum;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for graph vectors
	cudaStatus = hipMalloc((pDevicGraph->m_anOwnerId, nVerNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((pDevicGraph->m_anSelfConn, nVerNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((pDevicGraph->m_anVrtexSize, nVerNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((pDevicGraph->m_anRowOffset, (nVerNum + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((pDevicGraph->m_anNeighborId, nEdgeNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((pDevicGraph->m_anBetweenConn, nEdgeNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<<1, size>>>(pDevicGraph);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	// TODO

Error:
	hipFree(pDevicGraph->m_anBetweenConn);
	hipFree(pDevicGraph->m_anNeighborId);
	hipFree(pDevicGraph->m_anOwnerId);
	hipFree(pDevicGraph->m_anRowOffset);
	hipFree(pDevicGraph->m_anSelfConn);
	hipFree(pDevicGraph->m_anVrtexSize);

	return cudaStatus;
}