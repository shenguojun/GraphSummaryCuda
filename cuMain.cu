#include "hip/hip_runtime.h"
#include "file_reader.h"
#include "graph.h"
#include "gputimer.h"
#include "init.h"
#include "util.h"

#include <iostream>
#include <stdio.h>

#define NUM_THREAD_PER_BLOCK 192

__global__ void compute(Graph* pd_Graph, int nPart){
	printf("Hello world! I'm a thread in block %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int main(int argc, char** argv){
	FileReader cFileReader;
	Graph* ph_Graph = new Graph();
	// copy data from file to CPU
	cFileReader.ReadGraph(ph_Graph);
	// copy data from CPU TO GPU
	Graph * pd_Graph = new Graph();
	cudaMemoryHostToDeviceMalloc(pd_Graph, ph_Graph);
	GpuTimer timer;
	timer.Start();
	// run GPU thread
	int nPart = 0;
	int nBlockPerGrid = Util::calculateBlockPerGrid(ph_Graph->m_nEdgeNum, NUM_THREAD_PER_BLOCK, nPart);
	compute<<<nBlockPerGrid, NUM_THREAD_PER_BLOCK>>>(pd_Graph, nPart);
	timer.Stop();
	// copy data from GPU to CPU
	// TODO
	printf("Time elapsed = %g ms\n", timer.Elapsed());
	// memory free
	hipFree(pd_Graph->m_anBetweenConn);
	hipFree(pd_Graph->m_anNeighborId);
	hipFree(pd_Graph->m_anOwnerId);
	hipFree(pd_Graph->m_anRowOffset);
	hipFree(pd_Graph->m_anSelfConn);
	hipFree(pd_Graph->m_anVrtexSize);

	return 0;
}